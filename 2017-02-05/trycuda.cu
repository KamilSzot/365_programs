
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

int I = 500;
int J = 500;
int K = 500;



__global__
void mul(int I, int J, int K, float *x, float *y, float *z)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(int q=index; q<I*K; q+=stride) {
    int i = q / K;
    int k = q % K;
    z[q] = 0.0f;
    for(int j=0; j<J; j++) {
      z[q] += x[i*J+j] * y[j*K+k];
    }
  }
}

int main(void)
{
  setbuf(stdout, NULL);
  printf("Start\n");
  float *x, *y, *z;
  hipMallocManaged(&x, I*J*sizeof(float));
  hipMallocManaged(&y, J*K*sizeof(float));
  hipMallocManaged(&z, I*K*sizeof(float));

  for(int i = 0; i < I*J; i++)
  {
    x[i] = 1.0f;
  }

  for(int i = 0; i < J*K; i++)
  {
    y[i] = 2.0f;
  }

  int blockSize = 1; //512;
  int numBlocks = 1; //min(65535, (I*K + blockSize - 1) / blockSize);

  printf("Number %f\n", x[0]);
  mul<<<numBlocks, blockSize>>>(I, J, K, x, y, z);
  printf("Number %f\n", x[0]);
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
      printf("kernel launch failed with error \"%s\".\n",
             hipGetErrorString(cudaerr));

  // for(int i = 0; i < N; i++)
  // {
  //     if(fabs(y[i] - 3.0f)>0.000000001) {
  //         printf("Wrong! %d %f", i, y[i]);
  //         break;
  //     }
  // }

  printf("Number %f\n", x[0]);
  hipFree(x);
  hipFree(y);
  hipFree(z);
}
